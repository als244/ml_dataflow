#include "hip/hip_runtime.h"
#include "nvidia_ops.h"

// Define constants for vectorization dimensions
#define ROPE_VEC_SIZE 8
#define ROPE_PAIRS (ROPE_VEC_SIZE / 2)
#define ROPE_MAX_HEAD_DIM 256
#define ROPE_BANKED_SMEM_COLS (ROPE_MAX_HEAD_DIM / 2 + NUM_BANKS - 1) / NUM_BANKS

/* * A union to convert between a float4 vector and an array of four 
 * __hip_bfloat162 ROPE_BWD_PAIRS. This facilitates efficient 16-byte memory 
 * operations while allowing easy access to individual data ROPE_BWD_PAIRS.
 */
 typedef union {
    float4 f4;
    __hip_bfloat162 bf162[ROPE_PAIRS];
} rope_f4_bf162_converter;

extern "C" __global__ void default_rope_fp32_kernel(int num_tokens, int model_dim, int head_dim, int num_kv_heads, int theta, int * seq_positions, float * X_q, float * X_k){
    
    int row_ind = blockIdx.x;
    if (row_ind >= num_tokens) {
        return;
    }

    int seq_pos = seq_positions[row_ind];

    // assert blockDim.x == head_dim / 2

    // each thread updates two positions within each head
    // for each head in row

    // e.g. if head_dim = 128, then blockDim.x = 64, and base_dim_in_head = 0, 2, 4, 6, 8, 10, ... 126
    int base_dim_in_head = 2 * threadIdx.x;

    float head_dim_frac = (float) base_dim_in_head / (float) head_dim;
    float angle = powf(theta, -1 * head_dim_frac);
    float cos_val = cosf(seq_pos * angle);
    float sin_val = sinf(seq_pos * angle);

    float x_even;
    float x_odd;

    float rope_even;
    float rope_odd;


    // first do the queries
     float * X_q_row = X_q + ((uint64_t)row_ind * (uint64_t) model_dim);

    // advance through each head
    // we set blockDim.x to be head_dim / 2, so we know  no threads will step on each other
    // and that all values in the row are covered
    for (int cur_dim = base_dim_in_head; cur_dim < model_dim; cur_dim += head_dim){
        x_even = X_q_row[cur_dim];
        x_odd = X_q_row[cur_dim + 1];

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        X_q_row[cur_dim] = rope_even;
        X_q_row[cur_dim + 1] = rope_odd;
    }

    // during recompute X_k is NULL
    if (!X_k){
        return;
    }

    
    int kv_dim = num_kv_heads * head_dim;
    float * X_k_row = X_k + ((uint64_t)row_ind * (uint64_t)kv_dim);

    // now do the keys
    for (int cur_dim = base_dim_in_head; cur_dim < kv_dim; cur_dim += head_dim){
        x_even = X_k_row[cur_dim];
        x_odd = X_k_row[cur_dim + 1];

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        
        X_k_row[cur_dim] = rope_even;
        X_k_row[cur_dim + 1] = rope_odd;
    }
}


extern "C" __global__ void default_rope_fp16_kernel(int num_tokens, int model_dim, int head_dim, int num_kv_heads, int theta, int * seq_positions, __half * X_q, __half * X_k){
    
    int row_ind = blockIdx.x;
    if (row_ind >= num_tokens) {
        return;
    }

    int seq_pos = seq_positions[row_ind];

    // assert blockDim.x == head_dim / 2

    // each thread updates two positions within each head
    // for each head in row

    // e.g. if head_dim = 128, then blockDim.x = 64, and base_dim_in_head = 0, 2, 4, 6, 8, 10, ... 126
    int base_dim_in_head = 2 * threadIdx.x;

    float head_dim_frac = (float) base_dim_in_head / (float) head_dim;
    float angle = powf(theta, -1 * head_dim_frac);
    float cos_val = cosf(seq_pos * angle);
    float sin_val = sinf(seq_pos * angle);

    float x_even;
    float x_odd;

    float rope_even;
    float rope_odd;


    // first do the queries
     __half * X_q_row = X_q + ((uint64_t)row_ind * (uint64_t) model_dim);

    // advance through each head
    // we set blockDim.x to be head_dim / 2, so we know  no threads will step on each other
    // and that all values in the row are covered
    for (int cur_dim = base_dim_in_head; cur_dim < model_dim; cur_dim += head_dim){
        x_even = X_q_row[cur_dim];
        x_odd = X_q_row[cur_dim + 1];

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        X_q_row[cur_dim] = rope_even;
        X_q_row[cur_dim + 1] = rope_odd;
    }

    // during recompute X_k is NULL
    if (!X_k){
        return;
    }

    
    int kv_dim = num_kv_heads * head_dim;
    __half * X_k_row = X_k + ((uint64_t)row_ind * (uint64_t)kv_dim);

    // now do the keys
    for (int cur_dim = base_dim_in_head; cur_dim < kv_dim; cur_dim += head_dim){
        x_even = __half2float(X_k_row[cur_dim]);
        x_odd = __half2float(X_k_row[cur_dim + 1]);

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        
        X_k_row[cur_dim] = __float2half(rope_even);
        X_k_row[cur_dim + 1] = __float2half(rope_odd);
    }
}

extern "C" __global__ void default_rope_bf16_kernel(
    int num_tokens,
    int model_dim,
    int head_dim,
    int num_kv_heads,
    int theta,
    int* __restrict__ seq_positions, // Can also be restricted
    __hip_bfloat16* __restrict__ X_q,
    __hip_bfloat16* __restrict__ X_k) {
    
    // Banked shared memory to prevent bank conflicts
    __shared__ float smem_cos[NUM_BANKS][ROPE_BANKED_SMEM_COLS];
    __shared__ float smem_sin[NUM_BANKS][ROPE_BANKED_SMEM_COLS];

    const int row_ind = blockIdx.x;
    if (row_ind >= num_tokens) {
        return;
    }

    const int seq_pos = seq_positions[row_ind];
    const int num_pairs = head_dim / 2;

    // --- Phase 1: Write to shared memory using the banked layout ---
    for (int i = threadIdx.x; i < num_pairs; i += blockDim.x) {
        const int bank = i % NUM_BANKS;
        const int col = i / NUM_BANKS;
        const float current_dim = (float)(2 * i);
        const float inv_freq = powf(theta, -current_dim / (float)head_dim);
        sincosf((float)seq_pos * inv_freq, &smem_sin[bank][col], &smem_cos[bank][col]);
    }
    __syncthreads();

    // --- Phase 2: Apply RoPE, reading from banked shared memory ---
    
    // Process Queries
    const int num_q_vectors = model_dim / ROPE_VEC_SIZE;
    __hip_bfloat16* X_q_row = X_q + (uint64_t)row_ind * model_dim;

    for (int vec_idx = threadIdx.x; vec_idx < num_q_vectors; vec_idx += blockDim.x) {
        const int vec_start_dim = vec_idx * ROPE_VEC_SIZE;
        const int base_dim_in_head = vec_start_dim % head_dim;

        rope_f4_bf162_converter data;
        data.f4 = *( (float4*)(&X_q_row[vec_start_dim]) );

        #pragma unroll
        for (int i = 0; i < ROPE_PAIRS; ++i) {
            const int pair_idx = (base_dim_in_head / 2) + i;
            // Map the pair_idx to the banked 2D layout to read
            const int bank = pair_idx % NUM_BANKS;
            const int col = pair_idx / NUM_BANKS;

            const float2 vals_fp32 = __bfloat1622float2(data.bf162[i]);
            const float cos_val = smem_cos[bank][col];
            const float sin_val = smem_sin[bank][col];

            const float rope_even = cos_val * vals_fp32.x - sin_val * vals_fp32.y;
            const float rope_odd = cos_val * vals_fp32.y + sin_val * vals_fp32.x;

            data.bf162[i] = __floats2bfloat162_rn(rope_even, rope_odd);
        }
        *( (float4*)(&X_q_row[vec_start_dim]) ) = data.f4;
    }

    // Process Keys
    if (!X_k) return;
    __syncthreads();

    const int kv_dim = num_kv_heads * head_dim;
    const int num_k_vectors = kv_dim / ROPE_VEC_SIZE;
    __hip_bfloat16* X_k_row = X_k + (uint64_t)row_ind * kv_dim;

    for (int vec_idx = threadIdx.x; vec_idx < num_k_vectors; vec_idx += blockDim.x) {
        const int vec_start_dim = vec_idx * ROPE_VEC_SIZE;
        const int base_dim_in_head = vec_start_dim % head_dim;

        rope_f4_bf162_converter data;
        data.f4 = *( (float4*)(&X_k_row[vec_start_dim]) );

        #pragma unroll
        for (int i = 0; i < ROPE_PAIRS; ++i) {
            const int pair_idx = (base_dim_in_head / 2) + i;
            const int bank = pair_idx % NUM_BANKS;
            const int col = pair_idx / NUM_BANKS;

            const float2 vals_fp32 = __bfloat1622float2(data.bf162[i]);
            const float cos_val = smem_cos[bank][col];
            const float sin_val = smem_sin[bank][col];
            
            const float rope_even = cos_val * vals_fp32.x - sin_val * vals_fp32.y;
            const float rope_odd = cos_val * vals_fp32.y + sin_val * vals_fp32.x;

            data.bf162[i] = __floats2bfloat162_rn(rope_even, rope_odd);
        }
        *( (float4*)(&X_k_row[vec_start_dim]) ) = data.f4;
    }
}


extern "C" __global__ void default_rope_fp8e4m3_kernel(int num_tokens, int model_dim, int head_dim, int num_kv_heads, int theta, int * seq_positions, __hip_fp8_e4m3_fnuz * X_q, __hip_fp8_e4m3_fnuz * X_k){
    
    int row_ind = blockIdx.x;
    if (row_ind >= num_tokens) {
        return;
    }

    int seq_pos = seq_positions[row_ind];

    // assert blockDim.x == head_dim / 2

    // each thread updates two positions within each head
    // for each head in row

    // e.g. if head_dim = 128, then blockDim.x = 64, and base_dim_in_head = 0, 2, 4, 6, 8, 10, ... 126
    int base_dim_in_head = 2 * threadIdx.x;

    float head_dim_frac = (float) base_dim_in_head / (float) head_dim;
    float angle = powf(theta, -1 * head_dim_frac);
    float cos_val = cosf(seq_pos * angle);
    float sin_val = sinf(seq_pos * angle);

    float x_even;
    float x_odd;

    float rope_even;
    float rope_odd;


    // first do the queries
     __hip_fp8_e4m3_fnuz * X_q_row = X_q + ((uint64_t)row_ind * (uint64_t) model_dim);

    // advance through each head
    // we set blockDim.x to be head_dim / 2, so we know  no threads will step on each other
    // and that all values in the row are covered
    for (int cur_dim = base_dim_in_head; cur_dim < model_dim; cur_dim += head_dim){
        x_even = float(X_q_row[cur_dim]);
        x_odd = float(X_q_row[cur_dim + 1]);

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        X_q_row[cur_dim] = __hip_fp8_e4m3_fnuz(rope_even);
        X_q_row[cur_dim + 1] = __hip_fp8_e4m3_fnuz(rope_odd);
    }

    // during recompute X_k is NULL
    if (!X_k){
        return;
    }
    
    int kv_dim = num_kv_heads * head_dim;
    __hip_fp8_e4m3_fnuz * X_k_row = X_k + ((uint64_t)row_ind * (uint64_t)kv_dim);

    // now do the keys
    for (int cur_dim = base_dim_in_head; cur_dim < kv_dim; cur_dim += head_dim){
        x_even = float(X_k_row[cur_dim]);
        x_odd = float(X_k_row[cur_dim + 1]);

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        
        X_k_row[cur_dim] = __hip_fp8_e4m3_fnuz(rope_even);
        X_k_row[cur_dim + 1] = __hip_fp8_e4m3_fnuz(rope_odd);
    }
}

extern "C" __global__ void default_rope_fp8e5m2_kernel(int num_tokens, int model_dim, int head_dim, int num_kv_heads, int theta, int * seq_positions, __hip_fp8_e5m2_fnuz * X_q, __hip_fp8_e5m2_fnuz * X_k){
    
    int row_ind = blockIdx.x;
    if (row_ind >= num_tokens) {
        return;
    }

    int seq_pos = seq_positions[row_ind];

    // assert blockDim.x == head_dim / 2

    // each thread updates two positions within each head
    // for each head in row

    // e.g. if head_dim = 128, then blockDim.x = 64, and base_dim_in_head = 0, 2, 4, 6, 8, 10, ... 126
    int base_dim_in_head = 2 * threadIdx.x;

    float head_dim_frac = (float) base_dim_in_head / (float) head_dim;
    float angle = powf(theta, -1 * head_dim_frac);
    float cos_val = cosf(seq_pos * angle);
    float sin_val = sinf(seq_pos * angle);

    float x_even;
    float x_odd;

    float rope_even;
    float rope_odd;


    // first do the queries
     __hip_fp8_e5m2_fnuz * X_q_row = X_q + ((uint64_t)row_ind * (uint64_t) model_dim);

    // advance through each head
    // we set blockDim.x to be head_dim / 2, so we know  no threads will step on each other
    // and that all values in the row are covered
    for (int cur_dim = base_dim_in_head; cur_dim < model_dim; cur_dim += head_dim){
        x_even = float(X_q_row[cur_dim]);
        x_odd = float(X_q_row[cur_dim + 1]);

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        X_q_row[cur_dim] = __hip_fp8_e5m2_fnuz(rope_even);
        X_q_row[cur_dim + 1] = __hip_fp8_e5m2_fnuz(rope_odd);
    }

    // during recompute X_k is NULL
    if (!X_k){
        return;
    }
    
    int kv_dim = num_kv_heads * head_dim;
    __hip_fp8_e5m2_fnuz * X_k_row = X_k + ((uint64_t)row_ind * (uint64_t)kv_dim);

    // now do the keys
    for (int cur_dim = base_dim_in_head; cur_dim < kv_dim; cur_dim += head_dim){
        x_even = float(X_k_row[cur_dim]);
        x_odd = float(X_k_row[cur_dim + 1]);

        rope_even = cos_val * x_even - sin_val * x_odd;
        rope_odd = cos_val * x_odd + sin_val * x_even;
        
        X_k_row[cur_dim] = __hip_fp8_e5m2_fnuz(rope_even);
        X_k_row[cur_dim + 1] = __hip_fp8_e5m2_fnuz(rope_odd);
    }
}