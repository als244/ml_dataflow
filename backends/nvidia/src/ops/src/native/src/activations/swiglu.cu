#include "hip/hip_runtime.h"
#include "nvidia_ops.h"


extern "C" __global__ void default_swiglu_fp32_kernel(int num_rows, int num_cols, float * x_w1, float * x_w3, float * out){

        int row_num = blockIdx.x;
        int thread_id = threadIdx.x;

        if (row_num < num_rows){
                float x_w1_val;
                float x_w3_val;
                float silu_x_w1;
                for (int d = thread_id; d < num_cols; d+=blockDim.x){
                        x_w1_val = x_w1[row_num * num_cols + d];
                        x_w3_val = x_w3[row_num * num_cols + d];

                        // overwrite contents in x_w1
                        silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));

                        // normally would set out to be x_w1...
                        out[row_num * num_cols + d] = silu_x_w1 * x_w3_val;
                }
        }
}

extern "C" __global__ void default_swiglu_fp16_kernel(int num_rows, int num_cols, __half * x_w1, __half * x_w3, __half * out){

        int row_num = blockIdx.x;
        int thread_id = threadIdx.x;

        if (row_num < num_rows){
                float x_w1_val;
                float x_w3_val;
                float silu_x_w1;
                for (int d = thread_id; d < num_cols; d+=blockDim.x){
                        x_w1_val = __half2float(x_w1[row_num * num_cols + d]);
                        x_w3_val = __half2float(x_w3[row_num * num_cols + d]);

                        // overwrite contents in x_w1
                        silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));

                        // normally would set out to be x_w1...
                        out[row_num * num_cols + d] = __float2half(silu_x_w1 * x_w3_val);
                }
        }
}

/**
 * Highly optimized SwiGLU activation kernel for bfloat16 data using wide, 64-bit loads.
 *
 * This version uses C-style casts instead of C++ reinterpret_cast for pointer type conversion.
 * The underlying logic and performance are identical to the C++ version.
 * It reinterprets the bfloat16 pointers to perform 64-bit memory transactions, fetching
 * four bfloat16 values per load. This maximizes memory bandwidth utilization.
 *
 * @param num_rows The number of rows in the input tensors.
 * @param num_cols The number of columns in the input tensors. This MUST be divisible by 4.
 * @param x_w1 Pointer to the first input tensor (global memory).
 * @param x_w3 Pointer to the second input tensor (global memory).
 * @param out Pointer to the output tensor (global memory).
 */
 extern "C" __global__ void default_swiglu_bf16_kernel(int num_rows, int num_cols, const __hip_bfloat16 * __restrict__ x_w1, const __hip_bfloat16 * __restrict__ x_w3, __hip_bfloat16 * __restrict__ out) {
        
        // This kernel requires the column dimension to be divisible by 4 to use
        // 64-bit (float2) vectorized loads.
        const int num_cols_vec = num_cols / 4;
    
        // Reinterpret the bfloat16 pointers as float2 pointers using C-style casts.
        // sizeof(float2) is 8 bytes, so this allows us to load/store four
        // bfloat16 values (4 * 2 = 8 bytes) in a single instruction.
        const float2 *x_w1_vec = (const float2*)(x_w1);
        const float2 *x_w3_vec = (const float2*)(x_w3);
        float2 *out_vec = (float2*)(out);
    
        // Each block processes one row.
        const int row_num = blockIdx.x;
    
        // Early exit for blocks outside the valid row range.
        if (row_num >= num_rows) {
            return;
        }
    
        // Grid-stride loop. Each thread processes multiple sets of 4 elements.
        for (int i = threadIdx.x; i < num_cols_vec; i += blockDim.x) {
            // Calculate the linear index for the 64-bit vectorized access.
            const int idx = row_num * num_cols_vec + i;
    
            // --- Step 1: Vectorized 64-bit Load ---
            // Perform a single 64-bit load to get four bfloat16 values for each input.
            const float2 x_w1_packed = x_w1_vec[idx];
            const float2 x_w3_packed = x_w3_vec[idx];
    
            // --- Step 2: Unpack and Compute ---
            // To compute, we must unpack the four bfloat16 values into floats.
            // We can reinterpret our loaded float2 (8 bytes) as an array of two
            // __nv_bfloat162s (2 * 4 bytes) using C-style casts.
            const __hip_bfloat162* x_w1_b162s = (const __hip_bfloat162*)(&x_w1_packed);
            const __hip_bfloat162* x_w3_b162s = (const __hip_bfloat162*)(&x_w3_packed);
    
            // Convert the two bfloat162s into two float2s for each input.
            const float2 x_w1_f2_0 = __bfloat1622float2(x_w1_b162s[0]); // Elements 0, 1
            const float2 x_w1_f2_1 = __bfloat1622float2(x_w1_b162s[1]); // Elements 2, 3
    
            const float2 x_w3_f2_0 = __bfloat1622float2(x_w3_b162s[0]);
            const float2 x_w3_f2_1 = __bfloat1622float2(x_w3_b162s[1]);
    
            // Apply SiLU and element-wise multiply for all four elements.
            const float silu_0 = x_w1_f2_0.x / (1.0f + __expf(-x_w1_f2_0.x));
            const float silu_1 = x_w1_f2_0.y / (1.0f + __expf(-x_w1_f2_0.y));
            const float silu_2 = x_w1_f2_1.x / (1.0f + __expf(-x_w1_f2_1.x));
            const float silu_3 = x_w1_f2_1.y / (1.0f + __expf(-x_w1_f2_1.y));
    
            const float out_0 = silu_0 * x_w3_f2_0.x;
            const float out_1 = silu_1 * x_w3_f2_0.y;
            const float out_2 = silu_2 * x_w3_f2_1.x;
            const float out_3 = silu_3 * x_w3_f2_1.y;
    
            // --- Step 3: Pack and Vectorized 64-bit Store ---
            // Convert the four float results back into two bfloat162s.
            const __hip_bfloat162 out_b162_0 = __float22bfloat162_rn(make_float2(out_0, out_1));
            const __hip_bfloat162 out_b162_1 = __float22bfloat162_rn(make_float2(out_2, out_3));
    
            // Pack the two bfloat162s back into a single float2 for the 64-bit store.
            float2 out_packed;
            ((__hip_bfloat162*)(&out_packed))[0] = out_b162_0;
            ((__hip_bfloat162*)(&out_packed))[1] = out_b162_1;
    
            // Perform a single 64-bit store to write four bfloat16 results.
            out_vec[idx] = out_packed;
        }
}

extern "C" __global__ void naive_default_swiglu_bf16_kernel(int num_rows, int num_cols, __hip_bfloat16 * x_w1, __hip_bfloat16 * x_w3, __hip_bfloat16 * out){

        int row_num = blockIdx.x;
        int thread_id = threadIdx.x;

        if (row_num < num_rows){
                float x_w1_val;
                float x_w3_val;
                float silu_x_w1;
                for (int d = thread_id; d < num_cols; d+=blockDim.x){
                        x_w1_val = __bfloat162float(x_w1[row_num * num_cols + d]);
                        x_w3_val = __bfloat162float(x_w3[row_num * num_cols + d]);

                        // overwrite contents in x_w1
                        silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));

                        // normally would set out to be x_w1...
                        out[row_num * num_cols + d] = __float2bfloat16(silu_x_w1 * x_w3_val);
                }
        }
}

extern "C" __global__ void default_swiglu_fp8e4m3_kernel(int num_rows, int num_cols, __hip_fp8_e4m3_fnuz * x_w1, __hip_fp8_e4m3_fnuz * x_w3, __hip_fp8_e4m3_fnuz * out){

        int row_num = blockIdx.x;
        int thread_id = threadIdx.x;

        if (row_num < num_rows){
                float x_w1_val;
                float x_w3_val;
                float silu_x_w1;
                for (int d = thread_id; d < num_cols; d+=blockDim.x){
                        x_w1_val = float(x_w1[row_num * num_cols + d]);
                        x_w3_val = float(x_w3[row_num * num_cols + d]);

                        // overwrite contents in x_w1
                        silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));

                        // normally would set out to be x_w1...
                        out[row_num * num_cols + d] = __hip_fp8_e4m3_fnuz(silu_x_w1 * x_w3_val);
                }
        }
}

extern "C" __global__ void default_swiglu_fp8e5m2_kernel(int num_rows, int num_cols, __hip_fp8_e5m2_fnuz * x_w1, __hip_fp8_e5m2_fnuz * x_w3, __hip_fp8_e5m2_fnuz * out){

        int row_num = blockIdx.x;
        int thread_id = threadIdx.x;

        if (row_num < num_rows){
                float x_w1_val;
                float x_w3_val;
                float silu_x_w1;
                for (int d = thread_id; d < num_cols; d+=blockDim.x){
                        x_w1_val = float(x_w1[row_num * num_cols + d]);
                        x_w3_val = float(x_w3[row_num * num_cols + d]);

                        // overwrite contents in x_w1
                        silu_x_w1 = x_w1_val / (1 + expf(-1 * x_w1_val));

                        // normally would set out to be x_w1...
                        out[row_num * num_cols + d] = __hip_fp8_e5m2_fnuz(silu_x_w1 * x_w3_val);
                }
        }
}


