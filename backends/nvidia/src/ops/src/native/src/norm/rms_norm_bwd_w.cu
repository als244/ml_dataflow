#include "hip/hip_runtime.h"
#include "nvidia_ops.h"


extern "C" __global__ void rms_norm_bwd_w_fp32_fp32_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, float * X_inp, float * upstream_dX, float * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];


	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols);

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}


	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += upstream_dX[cur_row * n_cols + cur_dim_offset] * X_inp[cur_row * n_cols + cur_dim_offset] * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, weight_derivs[dim]);
	}
}



// Because X_inp is in row-major order we should be clever about doing column-wise dot products...

// at the end will do atomicAdds to dW because other blocks will have partial dot products as well

// cannot launch with more threads and n_cols otherwise will be bugs
// # blocks launched is a performance optimization and might be better with less due to less atomicAdds...
// definitely shouldn't launch with more than n_rows
extern "C" __global__ void rms_norm_bwd_w_fp16_fp16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __half * X_inp, __half * upstream_dX, __half * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];


	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols);

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __half2float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * __half2float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2half(weight_derivs[dim]));
	}
}


extern "C" __global__ void rms_norm_bwd_w_bf16_bf16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __hip_bfloat16 * X_inp, __hip_bfloat16 * upstream_dX, __hip_bfloat16 * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];


	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols);


	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __bfloat162float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * __bfloat162float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2bfloat16(weight_derivs[dim]));
	}
}

extern "C" __global__ void rms_norm_bwd_w_fp8e4m3_fp16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __hip_fp8_e4m3_fnuz * X_inp, __half * upstream_dX, __half * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols); 

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __half2float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2half(weight_derivs[dim]));
	}
}


extern "C" __global__ void rms_norm_bwd_w_fp8e4m3_bf16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __hip_fp8_e4m3_fnuz * X_inp, __hip_bfloat16 * upstream_dX, __hip_bfloat16 * dW) {
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols);

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __bfloat162float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2bfloat16(weight_derivs[dim]));
	}
}

extern "C" __global__ void rms_norm_bwd_w_fp8e5m2_fp16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __hip_fp8_e5m2_fnuz * X_inp, __half * upstream_dX, __half * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols); 

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __half2float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2half(weight_derivs[dim]));
	}
}


extern "C" __global__ void rms_norm_bwd_w_fp8e5m2_bf16_kernel(int n_rows, int n_cols, float eps, float * fwd_rms_vals, __hip_fp8_e5m2_fnuz * X_inp, __hip_bfloat16 * upstream_dX, __hip_bfloat16 * dW){
	
	// this gets dynamically allocated the size of model_dim
	extern __shared__ uint8_t sdata[];



	// working space when computing weight derivs...
	// the dot products will be updated here and when complete
	// will accumulate in dW

	// length equal to the number of columns
	float * weight_derivs = (float *) sdata; 

	// length should be equal to max number of rows per block
	// load in squared sums and then divide by n_cols and take sqrt
	float * recip_avgs = (float *) (weight_derivs + n_cols);

	int row_base = blockIdx.x;

	if (row_base >= n_rows){
		return;
	}

	int rows_per_block = n_rows / gridDim.x;
	
	int rows_remain = n_rows % gridDim.x;
	int row_offset;
	if (blockIdx.x < rows_remain){
		// this block will need to do an extra row
		rows_per_block += 1;
		// all prior blocks also had an extra row
		row_offset = row_base * rows_per_block;
	}
	else{
		row_offset = row_base * rows_per_block + rows_remain;
	}

	
	int thread_id = threadIdx.x;

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	

	// retrieve back the recip squared avgs
	// corresponding to this blocks rows
	for (int i = row_offset + thread_id; i < row_offset + rows_per_block; i+=blockDim.x){
		recip_avgs[i - row_offset] = fwd_rms_vals[i];
	}

	for (uint64_t i = thread_id; i < n_cols; i+=blockDim.x){
		weight_derivs[i] = 0;
	}

	__syncthreads();

	
	// ensure that # threads launched is less than n_cols
	int num_warps = blockDim.x / 32;
	int dims_per_warp = ceilf((float) n_cols / (float) num_warps);

	int warp_iter;
	int cur_dim_offset;

	float cur_recip_avg;

	for (int cur_row = row_offset; cur_row < row_offset + rows_per_block; cur_row++){

		cur_recip_avg = recip_avgs[cur_row - row_offset];

		// each warp within threadblock will have a different dim_offset
		// and only be respno
		warp_iter = 0;
		cur_dim_offset = dims_per_warp * warp_id + lane_id;
		while ((warp_iter * 32) < (dims_per_warp) && (cur_dim_offset < n_cols)){

			// portion of dot product to update weight at cur_dim_offset
			// because each warp has their own section of dims some can run ahead
			// vs. others and ensure that the shared memory weigth_derivs (portions of column-wise dot product)
			// are still OK...

			// apply chain rule by multiplying with the upstream value...
			weight_derivs[cur_dim_offset] += __bfloat162float(upstream_dX[cur_row * n_cols + cur_dim_offset]) * float(X_inp[cur_row * n_cols + cur_dim_offset]) * cur_recip_avg;
			cur_dim_offset += 32;
			warp_iter++;
		}
	}

	// ensure all warps finish their portion of block
	__syncthreads();

	// now need to do atomic add into the global dW for this section of rows
	for (uint64_t dim = thread_id; dim < n_cols; dim+=blockDim.x){
		atomicAdd(dW + dim, __float2bfloat16(weight_derivs[dim]));
	}
}