#include "hip/hip_runtime.h"
#include "nvidia_ops.h"


// TODO: vectorize this kernel, think about if block-per row is ideal, and handle smem more efficiently...
extern "C" __global__ void default_router_bwd_x_bf16_bf16_kernel(int num_tokens, int model_dim, int num_routed_experts, int top_k_active,
                                                int expert_id, int * expert_counts_cumsum, int * expert_mapping, 
                                                uint16_t * chosen_experts, float * token_expert_weights,
                                                __hip_bfloat16 * expert_out, __hip_bfloat16 * upstream_dX,
                                                __hip_bfloat16 * dX_routed,
                                                __hip_bfloat16 * dX_expert_out){


    int new_row_ind = blockIdx.x;

    if (new_row_ind >= num_tokens){
        return;
    }

    int expert_base_ind = expert_counts_cumsum[expert_id] - num_tokens;

    float thread_sum = 0;
    
    __shared__ float warp_sums[32];

    __shared__ float smem_token_weight;

      // could load the upstream_dX into shared memory for efficiency as we need to copy it to dX_expert_out...

    float token_weight;

    int orig_row_ind;

    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    while (new_row_ind < num_tokens){

        if (threadIdx.x < 32){
            warp_sums[threadIdx.x] = 0;
        }
        
        // 1.) get orig row ind corresponding to this token
        orig_row_ind = expert_mapping[expert_base_ind + new_row_ind];

        // 2.) compute dot product of expert_out and upstream dX

        thread_sum = 0;
        
        for (int i = threadIdx.x; i < model_dim; i += blockDim.x){
            thread_sum += __bfloat162float(expert_out[new_row_ind * model_dim + i]) * __bfloat162float(upstream_dX[orig_row_ind * model_dim + i]);
        }

        __syncwarp();

        for (int offset = 16; offset > 0; offset >>= 1) {
            thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, offset);
        }

        if (lane_id == 0) {
            warp_sums[warp_id] = thread_sum;
        }

        __syncthreads();

        // 3.) complete the dot-product and populate dX_routed with this scalar value

        if (warp_id == 0){
            thread_sum = warp_sums[lane_id];

            for (int offset = 16; offset > 0; offset >>= 1) {
                thread_sum += __shfl_down_sync(0xFFFFFFFF, thread_sum, offset);
            }

            if (lane_id == 0){
                dX_routed[orig_row_ind * num_routed_experts + expert_id] = thread_sum;
            }

            __syncwarp();

            // figure out the token_weight is
            for (int i = lane_id; i < top_k_active; i += 32){
                if (chosen_experts[orig_row_ind * top_k_active + i] == (uint16_t) expert_id){
                    smem_token_weight = token_expert_weights[orig_row_ind * top_k_active + i];
                    break;
                }
            }
            
        }

        __syncthreads();

        token_weight = smem_token_weight;

        if (token_weight == 0.0f){
            if (threadIdx.x == 0){
                printf("Error: token_weight is 0.0f for orig token %d and expert %d...\n", orig_row_ind, expert_id);
            }
            return;
        }

        // 4.) repopulate dX_expert_out with the rows from inp_grad_stream -> X * weight assoicated with this expert (for each token)...
        //     -- really should be loading upstream_dX into shared memory for efficiency...
        for (int i = threadIdx.x; i < model_dim; i += blockDim.x){
            dX_expert_out[new_row_ind * model_dim + i] = __float2bfloat16(__bfloat162float(upstream_dX[orig_row_ind * model_dim + i]) * token_weight);
        }

        new_row_ind += gridDim.x;
    }
}







