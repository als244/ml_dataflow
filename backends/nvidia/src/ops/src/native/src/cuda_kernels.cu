#include "hip/hip_runtime.h"

// Making all functions available within one cubin file...

#include "preprocess/embedding_table.cu"
#include "preprocess/embedding_table_bwd_w.cu"

#include "norm/rms_norm.cu"
#include "norm/rms_norm_recompute.cu"
#include "norm/rms_norm_bwd_x.cu"
#include "norm/rms_norm_bwd_w.cu"
#include "norm/rms_norm_bwd_w_combine.cu"
#include "norm/rms_norm_noscale.cu"
#include "norm/rms_norm_noscale_bwd_x.cu"

#include "attention_misc/rope.cu"
#include "attention_misc/rope_bwd_x.cu"
#include "attention_misc/copy_to_seq_context.cu"

#include "optimizer/adamw_step.cu"

#include "moe/select_experts.cu"

#include "activations/swiglu.cu"
#include "activations/swiglu_bwd_x.cu"

#include "loss_misc/softmax.cu"
#include "loss_misc/cross_entropy.cu"
