#include "hip/hip_runtime.h"
#include "nvidia_ops.h"

extern "C" __global__ void default_softmax_fp32_fp32_kernel(int n_rows, int n_cols, float * X, float * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	float * row_start = X + row_offset;
	float * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ float warp_maxs[32];
	__shared__ float warp_sums[32];
	__shared__ float global_max[1];
	__shared__ float global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	if (warp_id == 0){
		warp_maxs[lane_id] = CONST_DEV_FLOAT_NEG_INF;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	float other_val;

	float new_max = CONST_DEV_FLOAT_NEG_INF;

	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = max(new_max, row_start[cur_ind]);

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = max(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = max(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	float overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(row_start[cur_ind] - overall_max);

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = expf(row_start[cur_ind] - overall_max) / overall_sum;
		cur_ind += num_warps * 32;
	}
}


extern "C" __global__ void default_softmax_fp16_fp16_kernel(int n_rows, int n_cols, __half * X, __half * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__half * row_start = X + row_offset;
	__half * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __half warp_maxs[32];
	__shared__ __half warp_sums[32];
	__shared__ __half global_max[1];
	__shared__ __half global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_FP16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__half other_val;

	__half new_max = NEG_INF_DEV_FP16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, row_start[cur_ind]);

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__half overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__half2float(row_start[cur_ind] - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = __float2half(expf(__half2float(row_start[cur_ind] - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}

extern "C" __global__ void default_softmax_bf16_bf16_kernel(int n_rows, int n_cols, __hip_bfloat16 * X, __hip_bfloat16 * out) {
    // Each block processes one row
    uint64_t row_ind = blockIdx.x;
    if (row_ind >= n_rows) {
        return;
    }

    uint64_t row_offset = row_ind * ((uint64_t) n_cols);
    __hip_bfloat16* row_start = X + row_offset;
    __hip_bfloat16* out_row_start = out + row_offset;

    // Use float for intermediate computations for precision
    float thread_max = -FLT_MAX;
    float thread_sum = 0.0f;

    // =========================================================================
    // Pass 1: Online calculation of max and sum for each thread's data chunk
    // =========================================================================
    for (int i = threadIdx.x; i < n_cols; i += blockDim.x) {
        float val = __bfloat162float(row_start[i]);
        
        float old_max = thread_max;
        thread_max = fmaxf(thread_max, val);

		thread_sum = thread_sum * expf(old_max - thread_max) + expf(val - thread_max);
    }

    // =========================================================================
    // Block-wide reduction of (max, sum) pairs
    // =========================================================================
    
    // Step 1: Warp-level reduction using shuffle instructions
    unsigned mask = 0xFFFFFFFFU;
    for (int offset = 16; offset > 0; offset >>= 1) {
        // Get partner thread's (max, sum)
        float partner_max = __shfl_down_sync(mask, thread_max, offset);
        float partner_sum = __shfl_down_sync(mask, thread_sum, offset);

        // Combine the pairs using the same online logic
        float old_max = thread_max;
        thread_max = fmaxf(thread_max, partner_max);
        float scale = expf(old_max - thread_max);
        
        thread_sum = thread_sum * scale + partner_sum * expf(partner_max - thread_max);
    }

    // Step 2: Inter-warp reduction using shared memory
    // Each warp's lane 0 now holds the warp's (max, sum)
    __shared__ float smem_max[32]; // Max warps per block = 1024 / 32 = 32
    __shared__ float smem_sum[32];

    int warp_id = threadIdx.x / 32;
    int lane_id = threadIdx.x % 32;

    if (lane_id == 0) {
        smem_max[warp_id] = thread_max;
        smem_sum[warp_id] = thread_sum;
    }

    __syncthreads();

    // Step 3: Final reduction by the first warp
    // Load warp-reduced values into first warp's registers
    if (warp_id == 0) {
        // Only load if there's a valid corresponding warp
        if (lane_id * 32 < blockDim.x) {
            thread_max = smem_max[lane_id];
            thread_sum = smem_sum[lane_id];
        } else {
            thread_max = -FLT_MAX;
            thread_sum = 0.0f;
        }

        // Final reduction within the first warp
        for (int offset = 16; offset > 0; offset >>= 1) {
             float partner_max = __shfl_down_sync(mask, thread_max, offset);
             float partner_sum = __shfl_down_sync(mask, thread_sum, offset);
             
			 float old_max = thread_max;
             thread_max = fmaxf(thread_max, partner_max);
             float scale = expf(old_max - thread_max);
             thread_sum = thread_sum * scale + partner_sum * expf(partner_max - thread_max);
        }
    }

    // The final (max, sum) is in lane 0 of warp 0. Broadcast to all threads.
    if (warp_id == 0 && lane_id == 0) {
        smem_max[0] = thread_max;
        smem_sum[0] = thread_sum;
    }
    
    __syncthreads();
    
    float block_max = smem_max[0];
    float block_sum = smem_sum[0];

    // =========================================================================
    // Pass 2: Apply normalization and write output
    // =========================================================================
    for (int i = threadIdx.x; i < n_cols; i += blockDim.x) {
        float val = __bfloat162float(row_start[i]);
        float final_val = expf(val - block_max) / block_sum;
        out_row_start[i] = __float2bfloat16(final_val);
    }
}

extern "C" __global__ void naive_default_softmax_bf16_bf16_kernel(int n_rows, int n_cols, __hip_bfloat16 * X, __hip_bfloat16 * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__hip_bfloat16 * row_start = X + row_offset;
	__hip_bfloat16 * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __hip_bfloat16 warp_maxs[32];
	__shared__ __hip_bfloat16 warp_sums[32];
	__shared__ __hip_bfloat16 global_max[1];
	__shared__ __hip_bfloat16 global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;

	unsigned warp_mask = 0xFFFFFFFFU;
	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_BF16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__hip_bfloat16 other_val;

	__hip_bfloat16 new_max = NEG_INF_DEV_BF16;

    // get row max:

    for (int i = thread_id; i < n_cols; i += blockDim.x){
        new_max = __hmax(new_max, row_start[i]);
    }

	__syncwarp();

    // get warp max

    for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
        other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
        new_max = __hmax(new_max, other_val);
    }

    if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

    // get row max

	__syncthreads();

    if (warp_id == 0){
        new_max = warp_maxs[lane_id];

        for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
    }

    __syncthreads();

    __hip_bfloat16 overall_max = global_max[0];

    // get sum of exp(row - max)

    float new_sum = 0;

    for (int i = thread_id; i < n_cols; i += blockDim.x){
        new_sum += expf(__bfloat162float(row_start[i] - overall_max));
    }

	__syncwarp();

    // get warp sum

    for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
        new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
    }

    if (lane_id == 0){
		warp_sums[warp_id] = new_sum;
	}

	__syncthreads();

    // get block sum

    if (warp_id == 0){
        new_sum = warp_sums[lane_id];

        for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
            new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
        }

        if (lane_id == 0){
            global_sum[0] = new_sum;
        }
    }

    __syncthreads();

    float overall_sum = global_sum[0];

    // get output

    for (int i = thread_id; i < n_cols; i += blockDim.x){
        out_row_start[i] = __float2bfloat16(expf(__bfloat162float(row_start[i] - overall_max)) / overall_sum);
    }
}

extern "C" __global__ void default_softmax_fp8e4m3_fp16_kernel(int n_rows, int n_cols, __hip_fp8_e4m3_fnuz * X, __half * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__hip_fp8_e4m3_fnuz * row_start = X + row_offset;
	__half * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __half warp_maxs[32];
	__shared__ __half warp_sums[32];
	__shared__ __half global_max[1];
	__shared__ __half global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_FP16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__half other_val;

	__half new_max = NEG_INF_DEV_FP16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, __half(row_start[cur_ind]));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__half overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__half2float(__half(row_start[cur_ind]) - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = __float2half(expf(__half2float(__half(row_start[cur_ind]) - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}


extern "C" __global__ void default_softmax_fp8e4m3_bf16_kernel(int n_rows, int n_cols, __hip_fp8_e4m3_fnuz * X, __hip_bfloat16 * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__hip_fp8_e4m3_fnuz * row_start = X + row_offset;
	__hip_bfloat16 * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __hip_bfloat16 warp_maxs[32];
	__shared__ __hip_bfloat16 warp_sums[32];
	__shared__ __hip_bfloat16 global_max[1];
	__shared__ __hip_bfloat16 global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_BF16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__hip_bfloat16 other_val;

	__hip_bfloat16 new_max = NEG_INF_DEV_BF16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, __hip_bfloat16(row_start[cur_ind]));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__hip_bfloat16 overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__bfloat162float(__hip_bfloat16(row_start[cur_ind]) - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = __float2bfloat16(expf(__bfloat162float(__hip_bfloat16(row_start[cur_ind]) - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}


extern "C" __global__ void default_softmax_fp8e5m2_fp16_kernel(int n_rows, int n_cols, __hip_fp8_e5m2_fnuz * X, __half * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__hip_fp8_e5m2_fnuz * row_start = X + row_offset;
	__half * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __half warp_maxs[32];
	__shared__ __half warp_sums[32];
	__shared__ __half global_max[1];
	__shared__ __half global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_FP16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__half other_val;

	__half new_max = NEG_INF_DEV_FP16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, __half(row_start[cur_ind]));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__half overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__half2float(__half(row_start[cur_ind]) - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = __float2half(expf(__half2float(__half(row_start[cur_ind]) - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}


extern "C" __global__ void default_softmax_fp8e5m2_bf16_kernel(int n_rows, int n_cols, __hip_fp8_e5m2_fnuz * X, __hip_bfloat16 * out) {

	uint64_t row_ind = blockIdx.x;

	uint64_t row_offset = row_ind * ((uint64_t) n_cols);
	__hip_fp8_e5m2_fnuz * row_start = X + row_offset;
	__hip_bfloat16 * out_row_start = out + row_offset;

	int thread_id = threadIdx.x;

	__shared__ __hip_bfloat16 warp_maxs[32];
	__shared__ __hip_bfloat16 warp_sums[32];
	__shared__ __hip_bfloat16 global_max[1];
	__shared__ __hip_bfloat16 global_sum[1];

	int warp_id = thread_id / 32;
	int lane_id = thread_id % 32;
	int num_warps = blockDim.x / 32;

	

	if (warp_id == 0){
		warp_maxs[lane_id] = NEG_INF_DEV_BF16;
		warp_sums[lane_id] = 0;
	}

	__syncthreads();

	__hip_bfloat16 other_val;

	__hip_bfloat16 new_max = NEG_INF_DEV_BF16;


	unsigned warp_mask = 0xFFFFFFFFU;

	int cur_ind = thread_id;

	// Assuming N is a multiple of 32 for simplicity...
	while (cur_ind < n_cols){

		new_max = __hmax(new_max, __hip_bfloat16(row_start[cur_ind]));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_maxs[warp_id] = new_max;
	}

	__syncthreads();


	if (warp_id == 0){

		new_max = warp_maxs[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			other_val = __shfl_down_sync(warp_mask, new_max, warp_offset);
			new_max = __hmax(new_max, other_val);
		}

		if (lane_id == 0){
			global_max[0] = new_max;
		}
	}

	__syncthreads();


	// now do sums

	cur_ind = thread_id;

	__hip_bfloat16 overall_max = global_max[0];

	float total_sum = 0;
	float new_sum;
	while (cur_ind < n_cols){

		new_sum = expf(__bfloat162float(__hip_bfloat16(row_start[cur_ind]) - overall_max));

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			new_sum += __shfl_down_sync(warp_mask, new_sum, warp_offset);
		}

		if (lane_id == 0){
			total_sum += new_sum;
		}

		cur_ind += num_warps * 32;
	}

	if (lane_id == 0){
		warp_sums[warp_id] = total_sum;
	}

	__syncthreads();

	if (warp_id == 0){

		total_sum = warp_sums[lane_id];

		#pragma unroll
		for (int warp_offset = 16; warp_offset > 0; warp_offset >>= 1){
			total_sum += __shfl_down_sync(warp_mask, total_sum, warp_offset);
		}

		if (lane_id == 0){
			global_sum[0] = total_sum;
		}
	}

	__syncthreads();


	// now do output

	float overall_sum = global_sum[0];

	cur_ind = thread_id;

	while (cur_ind < n_cols){

		out_row_start[cur_ind] = __float2bfloat16(expf(__bfloat162float(__hip_bfloat16(row_start[cur_ind]) - overall_max)) / overall_sum);
		cur_ind += num_warps * 32;
	}
}
